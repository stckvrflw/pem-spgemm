#include "hip/hip_runtime.h"
/*
Author: Petrus E. Manurung
*/

#include <cstdlib>
#include <cstdio>

#include <fstream>
#include <thread>
#include <algorithm>
#include <chrono>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/scan.h>
#include <cooperative_groups/reduce.h>

#include <thrust/sort.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/unique.h>
#include <thrust/partition.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/scatter.h>

#include <cub/block/block_scan.cuh>
#include <cub/warp/warp_merge_sort.cuh>

#include <hipsparse.h>

#include <rmm/mr/device/cuda_async_memory_resource.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/device_vector.hpp>

#include "cx.h"
#include "fast_matrix_market/fast_matrix_market.hpp"
#include "TileCSR.h"
#include "utilities.h"

namespace cg = cooperative_groups;

template<typename ValueType>
void read_matrix_market
(
    char const *matrix_path,
    thrustHvecPin<int> &I, 
    thrustHvecPin<int> &J, 
    thrustHvecPin<ValueType> &vals, 
    int &rows, int &cols, int &nnz
)
{
    std::ifstream file(matrix_path);
    std::vector<int> _I;
    std::vector<int> _J;
    std::vector<ValueType> _vals;
    int _rows, _cols, _nnz;
    fast_matrix_market::read_matrix_market_triplet(file,
                                                    _rows, _cols,
                                                    _I, _J, _vals);
    _nnz = _vals.size();
    
    I.reserve(_nnz);
    J.reserve(_nnz);
    vals.reserve(_nnz);

    rows = _rows;
    cols = _cols;
    nnz = _nnz;

    I = _I;
    J = _J;
    vals = _vals;
    file.close();
}

template<unsigned tileSize = 16>
__global__ void __launch_bounds__(tileSize * tileSize)
decide_which_tile
(
    r_Ptr<long long> participating_tiles,
    cr_Ptr<int> d_I,
    cr_Ptr<int> d_J,
    int nnz
)
{
    auto grid = cg::this_grid();
    auto my_tid = grid.thread_rank();
    // out of range check
    if(my_tid >= nnz) return;
    int my_y = d_I[my_tid];
    int my_x = d_J[my_tid];
    // int this_tile_y = my_y / tileSize;
    // int this_tile_x = my_x / tileSize;
    int this_tile_y = my_y >> 4;
    int this_tile_x = my_x >> 4;

    // long long my_tile = (static_cast<long long>(this_tile_y) << 32) | this_tile_x;
    long long my_tile = this_tile_y;
    my_tile <<= 32;
    my_tile |= this_tile_x;
    participating_tiles[my_tid] = my_tile;
}

template<typename ValueType, int tileSize = 16>
__global__ void __launch_bounds__(tileSize * tileSize)
generate_tiles_csr
(
    r_Ptr<TileCSR_rev<ValueType, tileSize>> d_tiles,
    r_Ptr<ValueType> d_tiles_vals,
    r_Ptr<uint8_t> d_tiles_rowColIdx,
    cr_Ptr<long long> participating_tiles,
    cr_Ptr<int> participating_tiles_size,
    r_Ptr<int> perTileNnz,
    cr_Ptr<int> d_J,
    cr_Ptr<ValueType> d_vals,
    cr_Ptr<int> d_rowPtr,
    int d_rowPtr_size
)
{
    using MaskType = uint16_t;
    using IdxType = uint8_t;

    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<32>(block);
    auto tid = block.thread_rank();

    unsigned block_id = grid.block_rank();
    while(block_id < *participating_tiles_size) 
    {
        long long block_tile = participating_tiles[block_id];
        int block_tile_x = *(reinterpret_cast<int*>(&block_tile));
        int block_tile_y = *(reinterpret_cast<int*>(&block_tile)+1);

        int block_tile_offset_x = block_tile_x << 4;
        int block_tile_offset_y = block_tile_y << 4;
        int block_d_tiles_offset = block_id;

        ValueType thread_val {};
        int thread_J = -1;

        __shared__ int temp_buffer[16];

        auto my_row_group = cg::tiled_partition<16>(block); // swap 16 to tileSize
        
        int my_row_group_rowPtr_offset = block_tile_offset_y + my_row_group.meta_group_rank();
        // if(my_row_group_rowPtr_offset >= d_rowPtr_size) return;
        if(my_row_group_rowPtr_offset < d_rowPtr_size)
        {
            int my_row_group_rowPtr = d_rowPtr[my_row_group_rowPtr_offset];
            int my_row_group_rowSize = d_rowPtr[my_row_group_rowPtr_offset + 1] - d_rowPtr[my_row_group_rowPtr_offset];

            int thread_offset = binarySearch(&d_J[my_row_group_rowPtr], block_tile_offset_x + (int)my_row_group.thread_rank(), my_row_group_rowSize);
            if(thread_offset != -1){
                thread_offset += my_row_group_rowPtr;
                thread_J = d_J[thread_offset];
                thread_val = d_vals[thread_offset];
            }
        }
        my_row_group.sync();

        IdxType my_RowColIdx = (static_cast<IdxType>(my_row_group.meta_group_rank()) << 4) | (thread_J%16);
        unsigned my_row_nnz = __popc(my_row_group.ballot(thread_J != -1));
        MaskType my_row_mask = thread_J != -1 ? 1 : 0;

        my_row_mask <<= (thread_J%tileSize);
        // my_row_mask |=  my_row_group.shfl_down(my_row_mask, 8);
        // my_row_mask |=  my_row_group.shfl_down(my_row_mask, 4);
        // my_row_mask |=  my_row_group.shfl_down(my_row_mask, 2);
        // my_row_mask |=  my_row_group.shfl_down(my_row_mask, 1);
        my_row_mask = cg::reduce(my_row_group, my_row_mask, cg::bit_or<decltype(my_row_mask)>());

        if(my_row_group.thread_rank() == 0) {
            d_tiles[block_d_tiles_offset].mask[my_row_group.meta_group_rank()] = my_row_mask;
            temp_buffer[my_row_group.meta_group_rank()] = my_row_nnz;
        }
        block.sync();

        if(my_row_group.meta_group_rank() == 0) {
            my_row_nnz = temp_buffer[my_row_group.thread_rank()];
            my_row_nnz = cg::exclusive_scan(my_row_group, my_row_nnz);
            d_tiles[block_d_tiles_offset].rowPtr[my_row_group.thread_rank()] = my_row_nnz; 
        }

        int tile_offset = perTileNnz[block_id];

        if(block.thread_rank() == 0) {
            d_tiles[block_d_tiles_offset].vals = d_tiles_vals + tile_offset;
            d_tiles[block_d_tiles_offset].rowColIdx = d_tiles_rowColIdx + tile_offset;
        }
        block.sync();

        int my_loc = (thread_val != 0) ? 1 : 0;
        using BlockScan = hipcub::BlockScan<int, 256>;
        __shared__ typename BlockScan::TempStorage temp_storage;
        BlockScan(temp_storage).ExclusiveSum(my_loc, my_loc);

        if(thread_J != -1)
        {
            d_tiles[block_d_tiles_offset].vals[my_loc] = thread_val;
            d_tiles[block_d_tiles_offset].rowColIdx[my_loc] = my_RowColIdx;
        }

        block_id += grid.num_blocks();
    }
}

template<typename ValueType>
__global__ void __launch_bounds__(256)
__transpose_B_mask
(
    r_Ptr<uint16_t> Btiles_transposed_mask,
    cr_Ptr<TileCSR_rev<ValueType>> Btiles, 
    int Btiles_size
)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    uint16_t thread_buf[16];
    while(idx < Btiles_size)
    {
        uint16_t tile_mask[16];
        *(reinterpret_cast<ulonglong4*>(tile_mask)) = *(reinterpret_cast<ulonglong4 const*>(Btiles[idx].mask));
        
        #pragma unroll
        for(int m = 0; m < 16; ++m)
        {
            uint16_t temp = 0;
            #pragma unroll
            for(int i = 0; i < 16; ++i)
            {
                temp |= (((tile_mask[i] >> m) & 1) << i);
            }
            thread_buf[m] = temp;
        }
        *(reinterpret_cast<ulonglong4*>(&Btiles_transposed_mask[(idx*16)])) = *(reinterpret_cast<ulonglong4*>(thread_buf));

        idx += (gridDim.x << 8);
    }
}

__attribute__((optimize("O3")))
int cusparse_highLevelMultiply
(
    r_Ptr<int> dA_csrOffsets,
    r_Ptr<int> dA_columns,
    r_Ptr<float> dA_values,
    int A_num_rows,
    int A_num_cols,
    int A_nnz,
    r_Ptr<int> dB_csrOffsets,
    r_Ptr<int> dB_columns,
    r_Ptr<float> dB_values,
    int B_num_rows,
    int B_num_cols,
    int B_nnz,
    rmm::device_vector<int> *d_CtilePtr,
    rmm::device_vector<int> *d_CtileRowIdx,
    rmm::device_vector<int> *d_CtileColIdx,
    hipStream_t stream,
    rmm::exec_policy_nosync ASYNC_EXEC_POLICY
)
{
    int *dC_csrOffsets;
    int *dC_columns;
    int *dC_rows;
    float *dC_values;

    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA, matB, matC;

    hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType        computeType = HIP_R_32F;
    float               alpha       = 1.0f;
    float               beta        = 0.0f;

    void*  dBuffer1    = NULL, *dBuffer2   = NULL;
    size_t bufferSize1 = 0,    bufferSize2 = 0;

    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    CHECK_CUSPARSE( hipsparseSetStream(handle, stream) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, B_num_rows, B_num_cols, B_nnz,
                                      dB_csrOffsets, dB_columns, dB_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, 0,
                                      dC_csrOffsets, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )

    // allocate C offsets
    CHECK_CUDA( hipMallocAsync((void**) &dC_csrOffsets,
                           (A_num_rows + 1) * sizeof(int), stream) )
    // CHECK_CUDA( hipMalloc((void**) &dC_csrOffsets,
    //                        (A_num_rows + 1) * sizeof(int)) )

    // SpGEMM Computation
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )

    // ask bufferSize1 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, NULL) )
    CHECK_CUDA( hipMallocAsync((void**) &dBuffer1, bufferSize1, stream) )
    // CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )
    // inspect the matrices A and B to understand the memory requirement for
    // the next step
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, dBuffer1) )

    // ask bufferSize2 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, matA, matB, &beta, matC,
                               computeType, HIPSPARSE_SPGEMM_DEFAULT,
                               spgemmDesc, &bufferSize2, NULL) )
    CHECK_CUDA( hipMallocAsync((void**) &dBuffer2, bufferSize2, stream) )
    // CHECK_CUDA( hipMalloc((void**) &dBuffer2, bufferSize2) )

    // compute the intermediate product of A * B
    CHECK_CUSPARSE( hipsparseSpGEMM_compute(handle, opA, opB,
                                           &alpha, matA, matB, &beta, matC,
                                           computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDesc, &bufferSize2, dBuffer2) )
    // get matrix C non-zero entries C_nnz1
    int64_t C_num_rows1, C_num_cols1, C_nnz1;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1,
                                         &C_nnz1) )
    // allocate matrix C
    CHECK_CUDA( hipMallocAsync((void**) &dC_columns, C_nnz1 * sizeof(int), stream)   )
    CHECK_CUDA( hipMallocAsync((void**) &dC_values,  C_nnz1 * sizeof(float), stream) )
    CHECK_CUDA( hipMallocAsync((void**) &dC_rows, C_nnz1 * sizeof(int), stream) )
    // CHECK_CUDA( hipMalloc((void**) &dC_columns, C_nnz1 * sizeof(int))   )
    // CHECK_CUDA( hipMalloc((void**) &dC_values,  C_nnz1 * sizeof(float)) )

    // NOTE: if 'beta' != 0, the values of C must be update after the allocation
    //       of dC_values, and before the call of hipsparseSpGEMM_copy

    // update matC with the new pointers
    CHECK_CUSPARSE(
        hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values) )

    // if beta != 0, hipsparseSpGEMM_copy reuses/updates the values of dC_values

    // copy the final products to the matrix C
    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) )

    CHECK_CUSPARSE( hipsparseXcsr2coo(handle, dC_csrOffsets, C_nnz1, C_num_rows1, dC_rows, HIPSPARSE_INDEX_BASE_ZERO) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseSpGEMM_destroyDescr(spgemmDesc) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )

    auto _dCtile = thrust::device_pointer_cast(dC_csrOffsets);
    // auto _dCcols = thrust::device_pointer_cast(dC_columns);
    // auto _dCrows = thrust::device_pointer_cast(dC_rows);
    
    // yep, this is not wrong.
    // swapped because Xcsr2coo is confusing
    auto _dCcols = thrust::device_pointer_cast(dC_rows);
    auto _dCrows = thrust::device_pointer_cast(dC_columns);

    d_CtilePtr->resize(A_num_rows + 1);
    d_CtileRowIdx->resize(C_nnz1);
    d_CtileColIdx->resize(C_nnz1);
    thrust::copy(ASYNC_EXEC_POLICY, _dCtile, _dCtile + A_num_rows + 1, d_CtilePtr->begin());
    thrust::copy(ASYNC_EXEC_POLICY, _dCcols, _dCcols + C_nnz1, d_CtileColIdx->begin());
    thrust::copy(ASYNC_EXEC_POLICY, _dCrows, _dCrows + C_nnz1, d_CtileRowIdx->begin());

    hipFreeAsync(dC_csrOffsets, stream);
    hipFreeAsync(dC_columns, stream);
    hipFreeAsync(dC_values, stream);
    hipFreeAsync(dC_rows, stream);
    // hipFree(dC_csrOffsets);
    // hipFree(dC_columns);
    // hipFree(dC_values);

    return C_nnz1;
}

template<int pass>
__device__ __forceinline__
// we iterate on lane_iter, search on lane_targ
int __find_pairs
(
    r_Ptr<int> pairs_a,
    r_Ptr<int> pairs_b,
    r_Ptr<int> C_targetTile,
    int t_start,
    cr_Ptr<int> lane_iter,
    int lane_iter_len,
    cr_Ptr<int> lane_targ,
    int lane_targ_len,
    int iter_offset,
    int targ_offset,
    int AorB,
    cr_Ptr<int> B_tileOffsets,
    int insertion_start
)
{
    int __local_count = 0; // for use when pass = 0 only
    for(int i = 0; i < lane_iter_len; ++i) {
        int found = binarySearch(lane_targ, lane_iter[i], lane_targ_len);
        if(found != -1) {
            if constexpr(pass == 1) // 1 = second pass
            {   
                int first = iter_offset + i;
                int second = targ_offset + found;
                if(!AorB) // meaning its B, need to swap-- order is AB
                std::swap(first, second);
                second = B_tileOffsets[second];
                int targ_idx = insertion_start;
                pairs_a[targ_idx] = first;
                pairs_b[targ_idx] = second;
                C_targetTile[targ_idx] = t_start;
                ++insertion_start;
            }
            else // 0 = first pass, only count how many are there
            ++__local_count;
        }
    }
    return __local_count;
}

template<int pass>
__global__ void __launch_bounds__(256) 
search_pairs
(
    r_Ptr<int> pairs_a,
    r_Ptr<int> pairs_b,
    r_Ptr<int> C_targetTile,
    cr_Ptr<int> C_rowPtr,
    cr_Ptr<int> C_colIdx,
    cr_Ptr<int> A_rowPtr,
    cr_Ptr<int> A_colIdx,
    cr_Ptr<int> B_colPtr,
    cr_Ptr<int> B_rowIdx,
    int C_rowPtr_size,
    int C_colIdx_size,
    cr_Ptr<int> B_tileOffsets,
    cr_Ptr<int> C_rowIdx,
    int *__restrict__ pairs_counter,
    r_Ptr<int> pairs_insertion_offset
)
{
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();
    auto thread = cg::this_thread();

    int __first_pass_thread_counter = 0; // for pass = 0 only

    int t_start = grid.thread_rank();
    while(t_start < C_colIdx_size) {
        int t_C_col = C_colIdx[t_start];
        int t_C_row = C_rowIdx[t_start];
        decltype(A_colIdx) A_colIdx_segment = A_colIdx + A_rowPtr[t_C_row];
        decltype(B_rowIdx) B_rowIdx_segment = B_rowIdx + B_colPtr[t_C_col];
        int A_colIdx_segment_len = A_rowPtr[t_C_row + 1] - A_rowPtr[t_C_row];
        int B_rowIdx_segment_len = B_colPtr[t_C_col + 1] - B_colPtr[t_C_col];
        int AorB = A_colIdx_segment_len <= B_rowIdx_segment_len ? 1 : 0; // A = 1; B = 0;

        if constexpr(pass == 0) 
        {
        int curr_count = 0;
        if(AorB)
        curr_count = __find_pairs<0>(nullptr, nullptr, C_targetTile, t_start, A_colIdx_segment, A_colIdx_segment_len, B_rowIdx_segment, B_rowIdx_segment_len, A_rowPtr[t_C_row], B_colPtr[t_C_col], AorB, B_tileOffsets, 0);
        else
        curr_count = __find_pairs<0>(nullptr, nullptr, C_targetTile, t_start, B_rowIdx_segment, B_rowIdx_segment_len, A_colIdx_segment, A_colIdx_segment_len, B_colPtr[t_C_col], A_rowPtr[t_C_row], AorB, B_tileOffsets, 0);
        
        pairs_insertion_offset[t_start] = curr_count;
        __first_pass_thread_counter += curr_count;
        }
        else 
        {
        if(AorB)
        __find_pairs<1>(pairs_a, pairs_b, C_targetTile, t_start, A_colIdx_segment, A_colIdx_segment_len, B_rowIdx_segment, B_rowIdx_segment_len, A_rowPtr[t_C_row], B_colPtr[t_C_col], AorB, B_tileOffsets, pairs_insertion_offset[t_start]);
        else
        __find_pairs<1>(pairs_a, pairs_b, C_targetTile, t_start, B_rowIdx_segment, B_rowIdx_segment_len, A_colIdx_segment, A_colIdx_segment_len, B_colPtr[t_C_col], A_rowPtr[t_C_row], AorB, B_tileOffsets, pairs_insertion_offset[t_start]);
        }
        
        t_start += grid.num_threads();
    }

    if constexpr(pass == 0) 
    {
    block.sync();
    __shared__ int total;
    auto warp = cg::tiled_partition<32>(block);
    if(block.thread_rank() == 0) total = 0;
    block.sync();
    int warp_total = cg::reduce(warp, __first_pass_thread_counter, cg::plus<int>());
    if(warp.thread_rank() == 0) atomicAdd_block(&total, warp_total);
    block.sync();
    if(block.thread_rank() == 0) atomicAdd(pairs_counter, total);
    }
}

enum TAGS {
    // ONE,
    // SIXTEEN,
    NONDENSE,
    DENSE,
    TAGS_COUNT
};

__global__ void __launch_bounds__(256)
tag_pairs
(
    r_Ptr<uint8_t> d_pairs_tag,
    int d_pairs_tag_size,
    cr_Ptr<long long> d_pairs,
    cr_Ptr<int> C_targetTiles,
    cr_Ptr<int> C_pertileNnz,
    cr_Ptr<int> A_perTileNnz,
    cr_Ptr<int> B_perTileNnz
)
{
    auto compare = [](int A_nnz, int B_nnz) __attribute__((always_inline))
    {
        int m = max(A_nnz, B_nnz);
        if(m <= 16) return 0;
        if(m <= 32) return 1;
        if(m <= 64) return 2;
        if(m <= 128) return 3;
        if(m <= 192) return 4;
        return 5; // treat as dense
    };

    auto compare2 = [](int C_nnz) __attribute__((always_inline))
    {
        // if(C_nnz <= 16) return 0;
        // return 1;
        if(C_nnz <= 192) return TAGS::NONDENSE;
        // else if(C_nnz <= 16) return TAGS::SIXTEEN;
        return TAGS::DENSE;
    };

    int thread_start = cg::this_grid().thread_rank();
    while(thread_start < d_pairs_tag_size)
    {
        // int a = (d_pairs[thread_start] >> 32);
        // int b = (d_pairs[thread_start] & 0xFFFFFFFF);
        // int curr_A_perTileNnz = A_perTileNnz[a+1] - A_perTileNnz[a];
        // int curr_B_perTileNnz = B_perTileNnz[b+1] - B_perTileNnz[b];

        // d_pairs_tag[thread_start] = compare(curr_A_perTileNnz, curr_B_perTileNnz);

        d_pairs_tag[thread_start] = compare2(C_pertileNnz[C_targetTiles[thread_start]+1]-C_pertileNnz[C_targetTiles[thread_start]]);
        thread_start += cg::this_grid().num_threads();
    }
}

template<typename ValueType, int tileSize = 16>
__global__ void __launch_bounds__(4 * 32) 
allocate_C
(
    cr_Ptr<int> d_pairs_a,
    cr_Ptr<int> d_pairs_b,
    int d_pairs_size,

    r_Ptr<TileCSR_C_rev<ValueType, tileSize>> Ctiles,
    int Ctiles_size,
    r_Ptr<int> _C_perTileNnz,
    cr_Ptr<int> C_targetTiles,

    cr_Ptr<TileCSR_rev<ValueType,tileSize>> Atiles,
    cr_Ptr<TileCSR_rev<ValueType,tileSize>> Btiles,
    cr_Ptr<uint16_t> Btiles_transposed_mask
)
{
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<32>(block);
    
    auto local_group = cg::tiled_partition<16>(warp);
    int lgmgr = local_group.meta_group_rank();
    int lgtr = local_group.thread_rank();
    auto isZero = [](unsigned n) __attribute__((always_inline)) { return ((n | (~n + 1)) >> 31) & 1; };

    int quarter_mgr = lgtr >> 3;
    int quarter_tr = lgtr % 8;
    int quarter_local_group_pairs_idx_start = grid.block_rank() * 16 + (threadIdx.x >> 3);
    while(quarter_local_group_pairs_idx_start < d_pairs_size) 
    {
        int quarter_local_group_tile_idx_A = d_pairs_a[quarter_local_group_pairs_idx_start];
        int quarter_local_group_tile_idx_B = d_pairs_b[quarter_local_group_pairs_idx_start];

        unsigned C_mask = 0;
        #pragma unroll
        for(int n = 0; n < tileSize; ++n) C_mask |= (isZero((Atiles[quarter_local_group_tile_idx_A].mask[(quarter_tr<<1)] & Btiles_transposed_mask[(quarter_local_group_tile_idx_B<<4)+n])) << n);
        C_mask <<= 16;
        #pragma unroll
        for(int n = 0; n < tileSize; ++n) C_mask |= (isZero((Atiles[quarter_local_group_tile_idx_A].mask[(quarter_tr<<1)+1] & Btiles_transposed_mask[(quarter_local_group_tile_idx_B<<4)+n])) << n);

        TileCSR_C_rev<ValueType> *tileC = Ctiles + C_targetTiles[quarter_local_group_pairs_idx_start];
        atomicOr(&tileC->mask[quarter_tr], C_mask);

        quarter_local_group_pairs_idx_start += (grid.num_blocks() << 4);
    }

    grid.sync();

    int local_group_Ctiles_idx_start = grid.block_rank() * 4 * 2 + warp.meta_group_rank() * 2 + lgmgr;
    while(local_group_Ctiles_idx_start < Ctiles_size) 
    {
        unsigned my_Cmask = Ctiles[local_group_Ctiles_idx_start].mask[lgtr / 2];
        my_Cmask >>= (16 * ((lgtr % 2) ^ 0x1));
        my_Cmask &= 0xFFFF;
        int nnz = __popc(my_Cmask);
        int row_nnz = cg::exclusive_scan(local_group, nnz);
        Ctiles[local_group_Ctiles_idx_start].rowPtr[lgtr] = row_nnz;
        if(lgtr == 15) _C_perTileNnz[local_group_Ctiles_idx_start] = nnz + row_nnz;
        local_group.sync();

        local_group_Ctiles_idx_start += grid.num_blocks() * block.num_threads() / local_group.size();
    }
}

template<typename ValueType, int tileSize = 16>
__global__ void __launch_bounds__(4 * 32) 
C_setOffsets
(
    r_Ptr<TileCSR_C_rev<ValueType, tileSize>> Ctiles,
    int Ctiles_size,
    r_Ptr<int> _C_perTileNnz,
    
    r_Ptr<ValueType> Ctiles_vals,
    r_Ptr<uint8_t> Ctiles_rowColIdx
)
{
    auto grid = cg::this_grid();
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<32>(block);

    auto local_group = cg::tiled_partition<16>(warp);
    int lgmgr = local_group.meta_group_rank();
    int lgtr = local_group.thread_rank();

    int local_group_Ctiles_idx_start = (grid.block_rank() << 3) + (warp.meta_group_rank() << 1) + lgmgr;
    while(local_group_Ctiles_idx_start < Ctiles_size) 
    {
        unsigned my_Cmask = Ctiles[local_group_Ctiles_idx_start].mask[lgtr / 2];
        my_Cmask >>= (16 * ((lgtr % 2) ^ 0x1));
        my_Cmask &= 0xFFFF;
        cg::invoke_one(local_group, [&]{
            int Ctile_offset = _C_perTileNnz[local_group_Ctiles_idx_start];
            Ctiles[local_group_Ctiles_idx_start].rowColIdx = Ctiles_rowColIdx + Ctile_offset;
            Ctiles[local_group_Ctiles_idx_start].vals = Ctiles_vals + Ctile_offset;
        });

        int my_offset = cg::exclusive_scan(local_group, __popc(my_Cmask));

        if(__popc(my_Cmask) > 0) 
        {
            for(int n = 1; n <= __popc(my_Cmask); ++n)
            {
                unsigned c = __fns(my_Cmask, 0, n);
                unsigned my_rowColIdx = (lgtr << 4) | c;
                Ctiles[local_group_Ctiles_idx_start].rowColIdx[my_offset++] = my_rowColIdx;
            }
        }

        local_group_Ctiles_idx_start += grid.num_blocks() * block.num_threads() / local_group.size();
    }
}

template<typename ValueType, int tileSize = 16>
__global__ void 
__launch_bounds__(tileSize * tileSize / 2)
multiply_pairs_register_cache // lte 16
(
    cr_Ptr<long long> d_pairs,
    int d_pairs_size,
    
    r_Ptr<TileCSR_C_rev<ValueType, tileSize>> Ctiles,
    r_Ptr<int> _C_perTileNnz,
    cr_Ptr<int> C_targetTiles,

    cr_Ptr<TileCSR_rev<ValueType,tileSize>> Atiles,
    cr_Ptr<int> _A_perTileNnz,

    cr_Ptr<TileCSR_rev<ValueType,tileSize>> Btiles,
    cr_Ptr<int> _B_perTileNnz
)
{
    using ValType = double;
    __shared__ ValType perWarp_buffer[4 * 32];
    // __shared__ unsigned warp_mask_cache[4];

    // int volatile warp_tr = threadIdx.x % 32;
    int volatile warp_mgr = threadIdx.x >> 5;
    // int volatile halfwarp_mgr = warp_tr >> 4;
    int volatile halfwarp_mgr = (threadIdx.x%32)>>4;
    // int volatile halfwarp_tr = warp_tr % 16;
    int volatile halfwarp_tr = (threadIdx.x%32)%16;
    // auto volatile halfwarp_tiles = (halfwarp_mgr == 0) ? Atiles : Btiles;

    // perWarp_buffer[threadIdx.x] = 0;

    int warp_pairs_idx_start = (blockIdx.x << 2) + warp_mgr;
    while(warp_pairs_idx_start < d_pairs_size) 
    {
        int volatile halfwarp_tile_idx = *(reinterpret_cast<int const*>(&d_pairs[warp_pairs_idx_start]) + (halfwarp_mgr ^ 0x1));
        int volatile halfwarp_tile_nnz = (halfwarp_mgr == 0) 
                     ? (_A_perTileNnz[halfwarp_tile_idx+1] - _A_perTileNnz[halfwarp_tile_idx]) 
                     : (_B_perTileNnz[halfwarp_tile_idx+1] - _B_perTileNnz[halfwarp_tile_idx]);
        auto volatile halfwarp_tile = ((halfwarp_mgr == 0) ? Atiles : Btiles) + halfwarp_tile_idx;

        int volatile warp_tileC_idx = C_targetTiles[warp_pairs_idx_start];
        TileCSR_C_rev<ValueType> * volatile tileC = Ctiles + warp_tileC_idx;
        int volatile tileC_nnz = _C_perTileNnz[warp_tileC_idx + 1] - _C_perTileNnz[warp_tileC_idx];
  
        ValueType volatile my_val {};
        uint8_t volatile my_a {};
        uint8_t volatile my_b {};

        if(halfwarp_tr <= halfwarp_tile_nnz) 
        {
            my_val = halfwarp_tile->vals[halfwarp_tr];
            my_a = ((halfwarp_tile->rowColIdx[halfwarp_tr] >> ((halfwarp_mgr ^ 0x1)<<2)) & 0xFU);
            my_b = ((halfwarp_tile->rowColIdx[halfwarp_tr] >> ((halfwarp_mgr)<<2)) & 0xFU);
        }
        __syncwarp();
        
        int volatile C_idx = 0;
        while(C_idx < tileC_nnz)
        {
            uint8_t C_rowColIdx = tileC->rowColIdx[C_idx];

            if( my_val != 0 && my_a == ((C_rowColIdx >> ((halfwarp_mgr ^ 0x1)<<2)) & 0xFU) )
            {
                // auto coalesced = cg::coalesced_threads();

                unsigned match_mask = ((1 << my_b) << (halfwarp_mgr<<4));
                match_mask = cg::reduce(cg::coalesced_threads(), match_mask, cg::bit_or<unsigned>());
                match_mask = (match_mask >> 16) & (match_mask & 0xFFFF);
                if((1 << my_b) & match_mask)
                {
                    auto matched = cg::coalesced_threads();
                    int volatile n = matched.num_threads()>>1; // strange register usage per thread here without volatile(40->53)
                    if(n == 16) [[unlikely]]
                    {
                        ValueType row_sum = my_val * matched.shfl_down(my_val, 16);
                        row_sum += matched.shfl_down(row_sum, 1);            
                        row_sum += matched.shfl_down(row_sum, 2);      
                        row_sum += matched.shfl_down(row_sum, 4);      
                        row_sum += matched.shfl_down(row_sum, 8);      
                        if(matched.thread_rank() == 0) atomicAdd(&tileC->vals[C_idx], row_sum);
                    }
                    else [[likely]]
                    {
                        perWarp_buffer[(warp_mgr<<5)+matched.thread_rank()] = my_val;
                        matched.sync();
                        if(matched.thread_rank() == 0)
                        {
                            ValueType sum{};
                            for(int i = 0; i < n; ++i) sum += perWarp_buffer[(warp_mgr<<5)+i] * perWarp_buffer[(warp_mgr<<5)+n+i];
                            atomicAdd(&tileC->vals[C_idx], sum);
                        }
                    }
                }
            }
            __syncwarp();
            ++C_idx;
        }

        // warp_pairs_idx_start += gridDim.x * (blockDim.x >> 5);
        warp_pairs_idx_start += (gridDim.x << 2);
    }
}

template<typename ValueType, int tileSize = 16>
__global__ void 
__launch_bounds__(tileSize * tileSize / 2)
multiply_pairs_default
(
    cr_Ptr<int> d_pairs_a,
    cr_Ptr<int> d_pairs_b,
    
    r_Ptr<TileCSR_C_rev<ValueType, tileSize>> Ctiles,
    int Ctiles_size,
    r_Ptr<int> _C_perTileNnz,
    cr_Ptr<int> C_targetTiles,

    cr_Ptr<TileCSR_rev<ValueType,tileSize>> Atiles,
    cr_Ptr<TileCSR_rev<ValueType,tileSize>> Btiles,

    cr_Ptr<uint16_t> Btiles_transposed_mask,
    cr_Ptr<int> C_targetTiles_offset
)
{
    using IdxType = uint8_t;
    using MaskType = uint16_t;

    __shared__ MaskType warp_tile_mask      [4][2][16];
    __shared__ IdxType warp_tileC_rowColIdx [4][256];

    int volatile warp_tr = threadIdx.x % 32;
    int volatile warp_mgr = threadIdx.x >> 5;
    int volatile halfwarp_tr = (threadIdx.x%32)%16;
    int volatile halfwarp_mgr = (threadIdx.x%32)>>4;
    auto halfwarp_tile = halfwarp_mgr == 0 ? Atiles : Btiles;    
    auto halfwarp_pairs = halfwarp_mgr == 0 ? d_pairs_a : d_pairs_b;

    int warp_tileC_idx = (blockIdx.x << 2) + warp_mgr;
    while(warp_tileC_idx < Ctiles_size)
    {
        TileCSR_C_rev<ValueType> * volatile tileC = Ctiles + warp_tileC_idx;
        int volatile tileC_nnz = _C_perTileNnz[warp_tileC_idx + 1] - _C_perTileNnz[warp_tileC_idx];
        int volatile d_pairs_count = C_targetTiles_offset[warp_tileC_idx+1] - C_targetTiles_offset[warp_tileC_idx];

        for(int i = warp_tr; i < tileC_nnz; i += 32)
        {
            warp_tileC_rowColIdx[warp_mgr][i] = tileC->rowColIdx[i];
        }

        for(int pair = C_targetTiles_offset[warp_tileC_idx]; pair < C_targetTiles_offset[warp_tileC_idx] + d_pairs_count; ++pair) 
        {
            if(halfwarp_tr == 0)
            {
                *(reinterpret_cast<ulonglong4*>(&warp_tile_mask[warp_mgr][halfwarp_mgr])) 
                = 
                *(reinterpret_cast<ulonglong4 const*>(&halfwarp_tile[halfwarp_pairs[pair]].mask));
            }

            int A = d_pairs_a[pair];
            int B = d_pairs_b[pair];

            __syncwarp();

            // calculate C
            for(int n = warp_tr; n < tileC_nnz; n+=32)
            {
                ValueType sum = 0;
                int r = warp_tileC_rowColIdx[warp_mgr][n] >> 4;
                int c = warp_tileC_rowColIdx[warp_mgr][n] & 0xF;
                unsigned my_mask = warp_tile_mask[warp_mgr][0][r] & Btiles_transposed_mask[(B<<4)+c];
                while(my_mask)
                {
                    int A_offset = __popc( warp_tile_mask[warp_mgr][0][r] & (0xFFFFU >> (17-__ffs(my_mask))) );
                    int B_offset = __popc( warp_tile_mask[warp_mgr][1][__ffs(my_mask)-1] & (0xFFFFU >> (16-c)) );
                    sum += Atiles[A].vals[Atiles[A].rowPtr[r]+A_offset] * Btiles[B].vals[Btiles[B].rowPtr[__ffs(my_mask)-1]+B_offset];

                    my_mask &= (~(1 << (__ffs(my_mask)-1)));
                }
                tileC->vals[n] += sum;
            }
        }
        warp_tileC_idx += (gridDim.x << 2);
    }
}

template<typename ValueType, int tileSize = 16>
__global__ void 
__launch_bounds__(128)// 4 warps
multiply_pairs_tensor // gt 192
(
    cr_Ptr<long long> d_pairs,
    int d_pairs_size,
    
    r_Ptr<TileCSR_C_rev<ValueType, tileSize>> Ctiles,
    r_Ptr<int> _C_perTileNnz,
    cr_Ptr<int> C_targetTiles,

    cr_Ptr<TileCSR_rev<ValueType,tileSize>> Atiles,
    cr_Ptr<int> _A_perTileNnz,

    cr_Ptr<TileCSR_rev<ValueType,tileSize>> Btiles,
    cr_Ptr<int> _B_perTileNnz
)
{
    using ValType = double;
    __shared__ ValType tileA[16][16];
    __shared__ ValType tileB[16][16];
    __shared__ ValType tileC[16][16];

    auto block_copy_shmem_sync = [&](int A_idx, int B_idx) __attribute__((always_inline)){
        for(int n = threadIdx.x; n < _A_perTileNnz[A_idx+1]-_A_perTileNnz[A_idx]; ++n)
        {
            int r = Atiles[A_idx].rowColIdx[n] >> 4;
            int c = Atiles[A_idx].rowColIdx[n] & 0xF;
            tileA[r][c] = Atiles[A_idx].vals[n];
        }
        for(int n = threadIdx.x; n < _B_perTileNnz[B_idx+1]-_B_perTileNnz[B_idx]; ++n)
        {
            int r = Btiles[B_idx].rowColIdx[n] >> 4;
            int c = Btiles[B_idx].rowColIdx[n] & 0xF;
            tileB[r][c] = Btiles[B_idx].vals[n];
        }

        __syncthreads();
    };

    auto block_init_shmem_sync = [&] __attribute__((always_inline)) {
        #pragma unroll
        for(int n = threadIdx.x; n < 256; n+=128)
        {
            int shr = n>>4;
            int shc = n%16;
            tileC[shr][shc] = tileA[shr][shc] = tileB[shr][shc] = 0;
        }
        __syncthreads();
    };

    block_init_shmem_sync();

    int volatile tile = threadIdx.x >> 5;
    int volatile warp_tid = threadIdx.x % 32;
    int volatile block_pairs_idx_start = blockIdx.x;
    while(block_pairs_idx_start < d_pairs_size)
    {
        int A_idx = d_pairs[block_pairs_idx_start] >> 32;
        int B_idx = d_pairs[block_pairs_idx_start] & 0xFFFFFFFF;
        block_copy_shmem_sync(A_idx, B_idx);

        double a {}, b {};
        double2 c {};

        c.x = 0;
        c.y = 0;

        int row_offset = (tile >> 1) << 3;
        int col_offset = (tile % 2) << 3;

        if(tile%2 == 0){
            #pragma unroll
            for(int n = 0; n < (16/4); ++n)
            {
                a = tileA[row_offset+(warp_tid>>2)][(warp_tid%4)+(n<<2)];
                b = tileB[(warp_tid%4)+(n<<2)][col_offset+(warp_tid>>2)];

                asm volatile("mma.sync.aligned.m8n8k4.row.col.f64.f64.f64.f64 { %0, %1 }, {%2}, {%3}, { %4, %5 };" : "=d"(c.x), "=d"(c.y) : "d"(a), "d"(b), "d"(c.x), "d"(c.y));
            }
        }

        else
        {
            #pragma unroll
            for(int n = 3; n >= 0; --n)
            {
                a = tileA[row_offset+(warp_tid>>2)][(warp_tid%4)+(n<<2)];
                b = tileB[(warp_tid%4)+(n<<2)][col_offset+(warp_tid>>2)];

                asm volatile("mma.sync.aligned.m8n8k4.row.col.f64.f64.f64.f64 { %0, %1 }, {%2}, {%3}, { %4, %5 };" : "=d"(c.x), "=d"(c.y) : "d"(a), "d"(b), "d"(c.x), "d"(c.y));
            }
        }

        int row = warp_tid >> 2;
        int col1 = ((warp_tid % 4) << 1);
        int col2 = ((warp_tid % 4) << 1) + 1;

        // tileC[row_offset + row][col_offset + col1] = c.x;
        // tileC[row_offset + row][col_offset + col2] = c.y;
        // __syncthreads();

        int block_tileC_idx = C_targetTiles[block_pairs_idx_start];
        TileCSR_C_rev<ValueType> *our_tileC = Ctiles + block_tileC_idx;
        int tileC_nnz = _C_perTileNnz[block_tileC_idx + 1] - _C_perTileNnz[block_tileC_idx];

        // for(int t = threadIdx.x; t < tileC_nnz; t+=blockDim.x)
        // atomicAdd(&our_tileC->vals[t], tileC[(our_tileC->rowColIdx[t])>>4][(our_tileC->rowColIdx[t])&0xF]);

        block_init_shmem_sync();
        block_pairs_idx_start += gridDim.x;
    }
}

template<typename ValueType, int tileSize = 16>
__global__ void __launch_bounds__(tileSize * tileSize)
sanitize_C
(
    r_Ptr<int> rows, 
    r_Ptr<int> cols, 
    r_Ptr<ValueType> vals, 
    cr_Ptr<TileCSR_C_rev<ValueType>> Ctiles,
    int Ctiles_size,
    cr_Ptr<int> _C_rowPtr, 
    int _C_rowPtrSize,
    cr_Ptr<int> _C_tileRowIdx,
    cr_Ptr<int> _C_tileColIdx, 
    cr_Ptr<int> _C_perTile_Nnz
)
{
    auto warp = cg::tiled_partition<32>(cg::this_thread_block());

    int warp_Ctiles_id = (blockIdx.x << 3) + warp.meta_group_rank();
    while(warp_Ctiles_id < Ctiles_size)
    {
        int warp_Ctile_x = _C_tileColIdx[warp_Ctiles_id];
        int warp_Ctile_y = _C_tileRowIdx[warp_Ctiles_id];
        int warp_Ctile_offset = _C_perTile_Nnz[warp_Ctiles_id];
        for(int n = warp.thread_rank(); n < _C_perTile_Nnz[warp_Ctiles_id+1]-_C_perTile_Nnz[warp_Ctiles_id]; n+=warp.num_threads())
        {
            int idx = warp_Ctile_offset + n;
            uint8_t t_rowColIdx = Ctiles[warp_Ctiles_id].rowColIdx[n];
            ValueType t_val = Ctiles[warp_Ctiles_id].vals[n];
            rows[idx] = (warp_Ctile_y<<4) + (t_rowColIdx>>4);
            cols[idx] = (warp_Ctile_x<<4) + (t_rowColIdx&0xF);
            vals[idx] = t_val;
        }

        warp_Ctiles_id += (gridDim.x << 3);
    }
}

#define STREAM_A streams[STREAMA]
#define STREAM_B streams[STREAMB]
#define STREAM_C streams[STREAMC]
#define STREAM_D streams[STREAMD]
#define STREAM_E streams[STREAME]

enum STREAMS {
    STREAMA,
    STREAMB,
    STREAMC,
    STREAMD,
    STREAME,
    STREAMS_COUNT,
};

int main(int argc, char *argv[]) {
    if(argc <= 1 || argc > 3) {
        std::cout << "Provide matrix market file path for A and B (or TRANSPOSE for A * At). Exiting\n";
        exit(1);
    }

    constexpr int tileSize = 16;
    using ValueType = double;

    std::array<hipStream_t, STREAMS_COUNT> streams;
    std::for_each(streams.begin(), streams.end(), [](hipStream_t &s){hipStreamCreate(&s);});

    // HOST MATRIX A ----------------------
    thrustHvecPin<int> A_I, A_J;
    thrustHvecPin<ValueType> A_val;
    int A_rows, A_cols, A_nnz;
    //-------------------------------------

    // HOST MATRIX B ----------------------
    thrustHvecPin<int> B_I, B_J;
    thrustHvecPin<ValueType> B_val;
    int B_rows, B_cols, B_nnz;
    //-------------------------------------

    std::jthread read_A(read_matrix_market<ValueType>, std::ref(argv[1]), std::ref(A_I), std::ref(A_J), std::ref(A_val), std::ref(A_rows), std::ref(A_cols), std::ref(A_nnz));
    // read_matrix_market(argv[1], A_I, A_J, A_val, A_rows, A_cols, A_nnz);
    read_matrix_market(argv[1], B_I, B_J, B_val, B_rows, B_cols, B_nnz);
    read_A.join();

    std::cout 
    << "MATRIX A\n"
    << "filepath: " << argv[1] << "\n"
    << "Rows: " << A_rows << "\n"
    << "Cols: " << A_cols << "\n"
    << "Nnz: " << A_nnz << "\n";

    std::cout 
    << "MATRIX B\n"
    << "filepath: " << argv[1] << "\n"
    << "Rows: " << B_rows << "\n"
    << "Cols: " << B_cols << "\n"
    << "Nnz: " << B_nnz << "\n";

    int const OVERHEAD = (A_nnz + B_nnz) / 4;
    auto SPGEMM_MR = rmm::mr::cuda_async_memory_resource(sizeof(ValueType) * 2 * (A_nnz + B_nnz + OVERHEAD));
    auto SPGEMM_STREAM_ALLOCATOR_INT = [&SPGEMM_MR](hipStream_t STREAM) {return rmm::mr::thrust_allocator<int>(STREAM, SPGEMM_MR);};
    auto SPGEMM_STREAM_ALLOCATOR_FLOAT = [&SPGEMM_MR](hipStream_t STREAM) {return rmm::mr::thrust_allocator<float>(STREAM, SPGEMM_MR);};
    auto SPGEMM_STREAM_ALLOCATOR_LONGLONG = [&SPGEMM_MR](hipStream_t STREAM) {return rmm::mr::thrust_allocator<long long>(STREAM, SPGEMM_MR);};
    auto SPGEMM_STREAM_ALLOCATOR_VALUETYPE = [&SPGEMM_MR](hipStream_t STREAM) {return rmm::mr::thrust_allocator<ValueType>(STREAM, SPGEMM_MR);};
    auto SPGEMM_STREAM_ALLOCATOR_TILECSR_REV = [&SPGEMM_MR](hipStream_t STREAM) {return rmm::mr::thrust_allocator<TileCSR_rev<ValueType>>(STREAM, SPGEMM_MR);};
    auto SPGEMM_STREAM_ALLOCATOR_TILECSRC_REV = [&SPGEMM_MR](hipStream_t STREAM) {return rmm::mr::thrust_allocator<TileCSR_C_rev<ValueType>>(STREAM, SPGEMM_MR);};
    auto SPGEMM_STREAM_ALLOCATOR_UINT8 = [&SPGEMM_MR](hipStream_t STREAM) {return rmm::mr::thrust_allocator<uint8_t>(STREAM, SPGEMM_MR);};
    auto SPGEMM_STREAM_ALLOCATOR_UINT16 = [&SPGEMM_MR](hipStream_t STREAM) {return rmm::mr::thrust_allocator<uint16_t>(STREAM, SPGEMM_MR);};


    auto SPGEMM_TEMPORARY_MR = rmm::mr::cuda_async_memory_resource(sizeof(char) * OVERHEAD);
    auto ASYNC_EXEC_POLICY = [&SPGEMM_TEMPORARY_MR](auto STREAM){return rmm::exec_policy_nosync(STREAM, &SPGEMM_TEMPORARY_MR);};


    // DEVICE MATRIX A --------------------
    rmm::device_vector<int>         A_d_I(A_nnz, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_A));
    rmm::device_vector<int>         A_d_J(A_nnz, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_A));
    rmm::device_vector<ValueType>   A_d_val(A_nnz, SPGEMM_STREAM_ALLOCATOR_VALUETYPE(STREAM_A));
    //-------------------------------------

    // DEVICE MATRIX B --------------------
    rmm::device_vector<int>         B_d_I(B_nnz, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_B));
    rmm::device_vector<int>         B_d_J(B_nnz, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_B));
    rmm::device_vector<ValueType>   B_d_val(B_nnz, SPGEMM_STREAM_ALLOCATOR_VALUETYPE(STREAM_B));
    //-------------------------------------

    thrust::copy(ASYNC_EXEC_POLICY(STREAM_A), A_I.begin(), A_I.end(), A_d_I.begin());
    thrust::copy(ASYNC_EXEC_POLICY(STREAM_A), A_J.begin(), A_J.end(), A_d_J.begin());
    thrust::copy(ASYNC_EXEC_POLICY(STREAM_A), A_val.begin(), A_val.end(), A_d_val.begin());

    thrust::copy(ASYNC_EXEC_POLICY(STREAM_B), B_I.begin(), B_I.end(), B_d_I.begin());
    thrust::copy(ASYNC_EXEC_POLICY(STREAM_B), B_J.begin(), B_J.end(), B_d_J.begin());
    thrust::copy(ASYNC_EXEC_POLICY(STREAM_B), B_val.begin(), B_val.end(), B_d_val.begin());

    // if(argv[2] == "TRANSPOSE")
    // {
    //     auto zit = thrust::make_zip_iterator(thrust::make_tuple(B_d_J.begin(), B_d_I.begin(), B_val.begin()));
    //     thrust::sort(ASYNC_EXEC_POLICY(STREAM_B), zit, zit + B_nnz);
    // }

    int A_tileRows = (A_rows-1+tileSize) / tileSize;
    int A_tileCols = (A_cols-1+tileSize) / tileSize;
    int B_tileRows = (B_rows-1+tileSize) / tileSize;
    int B_tileCols = (B_cols-1+tileSize) / tileSize;
    rmm::device_vector<long long> A_participating_tiles(A_nnz, -1, SPGEMM_STREAM_ALLOCATOR_LONGLONG(STREAM_A));
    rmm::device_vector<long long> B_participating_tiles(B_nnz, -1, SPGEMM_STREAM_ALLOCATOR_LONGLONG(STREAM_B));

    dim3 A_threads_dwc{tileSize * tileSize};
    dim3 A_blocks_dwc{(A_nnz - 1 + A_threads_dwc.x)/A_threads_dwc.x};
    decide_which_tile<<<A_blocks_dwc, A_threads_dwc, 0, STREAM_A>>>
    (
        A_participating_tiles.data().get(), 
        A_d_I.data().get(), 
        A_d_J.data().get(), 
        A_nnz
    );
    dim3 B_threads_dwc{tileSize * tileSize};
    dim3 B_blocks_dwc{(B_nnz - 1 + B_threads_dwc.x)/B_threads_dwc.x};
    decide_which_tile<<<B_blocks_dwc, B_threads_dwc, 0, STREAM_B>>>
    (
        B_participating_tiles.data().get(), 
        B_d_I.data().get(), 
        B_d_J.data().get(), 
        B_nnz
    );
    
    rmm::device_vector<int> A_perTileNnz(SPGEMM_STREAM_ALLOCATOR_INT(STREAM_A));
    int cntA = 0;
    {
    thrust::sort(ASYNC_EXEC_POLICY(STREAM_A), A_participating_tiles.begin(), A_participating_tiles.end(), thrust::less<long long>());
    
    cntA = thrust::unique_count(ASYNC_EXEC_POLICY(STREAM_A), A_participating_tiles.begin(), A_participating_tiles.end());
    A_perTileNnz.resize(cntA + 1);
    thrust::reduce_by_key(ASYNC_EXEC_POLICY(STREAM_A), A_participating_tiles.begin(), A_participating_tiles.end(), thrust::make_constant_iterator<int>(1), thrust::make_discard_iterator(), A_perTileNnz.begin());
    thrust::exclusive_scan(ASYNC_EXEC_POLICY(STREAM_A),A_perTileNnz.begin(), A_perTileNnz.end(), A_perTileNnz.begin());

    auto newend = thrust::unique(ASYNC_EXEC_POLICY(STREAM_A), A_participating_tiles.begin(), A_participating_tiles.end());
    A_participating_tiles.erase(newend, A_participating_tiles.end());
    }

    rmm::device_vector<int> B_perTileNnz(SPGEMM_STREAM_ALLOCATOR_INT(STREAM_B));
    int cntB = 0;
    {
    thrust::sort(ASYNC_EXEC_POLICY(STREAM_B), B_participating_tiles.begin(), B_participating_tiles.end(), thrust::less<long long>());
    
    cntB = thrust::unique_count(ASYNC_EXEC_POLICY(STREAM_B), B_participating_tiles.begin(), B_participating_tiles.end());
    B_perTileNnz.resize(cntB + 1);
    thrust::reduce_by_key(ASYNC_EXEC_POLICY(STREAM_B), B_participating_tiles.begin(), B_participating_tiles.end(), thrust::make_constant_iterator<int>(1), thrust::make_discard_iterator(), B_perTileNnz.begin());
    thrust::exclusive_scan(ASYNC_EXEC_POLICY(STREAM_B), B_perTileNnz.begin(), B_perTileNnz.end(), B_perTileNnz.begin());

    auto newend = thrust::unique(ASYNC_EXEC_POLICY(STREAM_B), B_participating_tiles.begin(), B_participating_tiles.end());
    B_participating_tiles.erase(newend, B_participating_tiles.end());
    }

    rmm::device_vector<int> A_d_rowPtr(A_rows + 1, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_A));
    {
    auto zit = thrust::make_zip_iterator(thrust::make_tuple(A_d_I.begin(), A_d_J.begin(), A_d_val.begin()));
    thrust::stable_sort(ASYNC_EXEC_POLICY(STREAM_A), zit, zit + A_nnz);

    rmm::device_vector<int> A_d_rowPtr_tmp(A_rows, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_A));
    rmm::device_vector<int> A_d_index(A_rows, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_A));
    auto res = thrust::reduce_by_key(
        ASYNC_EXEC_POLICY(STREAM_A), 
        A_d_I.begin(), 
        A_d_I.end(), 
        thrust::make_constant_iterator<int>(1), 
        A_d_index.begin(), 
        A_d_rowPtr_tmp.begin());
    thrust::scatter(ASYNC_EXEC_POLICY(STREAM_A), A_d_rowPtr_tmp.begin(), res.second, A_d_index.begin(), A_d_rowPtr.begin());
    thrust::exclusive_scan(ASYNC_EXEC_POLICY(STREAM_A), A_d_rowPtr.begin(), A_d_rowPtr.end(), A_d_rowPtr.begin());
    }

    rmm::device_vector<int> B_d_rowPtr(B_rows + 1, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_B));
    {
    auto zit = thrust::make_zip_iterator(thrust::make_tuple(B_d_I.begin(), B_d_J.begin(), B_d_val.begin()));
    thrust::stable_sort(ASYNC_EXEC_POLICY(STREAM_B), zit, zit + B_nnz);

    rmm::device_vector<int> B_d_rowPtr_tmp(B_rows, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_B));
    rmm::device_vector<int> B_d_index(B_rows, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_B));
    auto res = thrust::reduce_by_key(
        ASYNC_EXEC_POLICY(STREAM_B), 
        B_d_I.begin(), 
        B_d_I.end(), 
        thrust::make_constant_iterator<int>(1), 
        B_d_index.begin(), 
        B_d_rowPtr_tmp.begin());
    thrust::scatter(ASYNC_EXEC_POLICY(STREAM_B), B_d_rowPtr_tmp.begin(), res.second, B_d_index.begin(), B_d_rowPtr.begin());
    thrust::exclusive_scan(ASYNC_EXEC_POLICY(STREAM_B), B_d_rowPtr.begin(), B_d_rowPtr.end(), B_d_rowPtr.begin());
    }

    hipEvent_t A_tileConversion_start, A_tileConversion_end, B_tileConversion_start, B_tileConversion_end;
    hipEventCreate(&A_tileConversion_start);
    hipEventCreate(&A_tileConversion_end);
    hipEventCreate(&B_tileConversion_start);
    hipEventCreate(&B_tileConversion_end);

    dim3 A_threads_gtc {tileSize * tileSize};
    dim3 A_blocks_gtc {cntA};

    rmm::device_vector<TileCSR_rev<ValueType>> Atiles(cntA, SPGEMM_STREAM_ALLOCATOR_TILECSR_REV(STREAM_A));
    rmm::device_vector<ValueType> Atiles_vals(A_nnz, SPGEMM_STREAM_ALLOCATOR_VALUETYPE(STREAM_A));
    rmm::device_vector<uint8_t> Atiles_rowColIdx(A_nnz, SPGEMM_STREAM_ALLOCATOR_UINT8(STREAM_A));
    rmm::device_vector<int> 
    A_d_cols(1, A_cols, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_A)), 
    A_participating_tiles_size(1, cntA, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_A));

    hipEventRecord(A_tileConversion_start, STREAM_A);
    generate_tiles_csr<ValueType><<<A_blocks_gtc, A_threads_gtc,0, STREAM_A>>>
    (
        Atiles.data().get(), 
        Atiles_vals.data().get(),
        Atiles_rowColIdx.data().get(),
        A_participating_tiles.data().get(), 
        A_participating_tiles_size.data().get(), 
        A_perTileNnz.data().get(), 
        A_d_J.data().get(), 
        A_d_val.data().get(), 
        A_d_rowPtr.data().get(),
        A_rows
    );
    hipEventRecord(A_tileConversion_end, STREAM_A);

    dim3 B_threads_gtc {tileSize * tileSize};
    dim3 B_blocks_gtc {cntB};

    rmm::device_vector<TileCSR_rev<ValueType>> Btiles(cntB, SPGEMM_STREAM_ALLOCATOR_TILECSR_REV(STREAM_B));
    rmm::device_vector<ValueType> Btiles_vals(B_nnz, SPGEMM_STREAM_ALLOCATOR_VALUETYPE(STREAM_B));
    rmm::device_vector<uint8_t> Btiles_rowColIdx(B_nnz, SPGEMM_STREAM_ALLOCATOR_UINT8(STREAM_B));
    rmm::device_vector<int> 
    B_d_cols(1, B_cols, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_B)), 
    B_participating_tiles_size(1, cntB, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_B));

    hipEventRecord(B_tileConversion_start, STREAM_B);
    generate_tiles_csr<ValueType><<<B_blocks_gtc, B_threads_gtc,0, STREAM_B>>>
    (
        Btiles.data().get(),
        Btiles_vals.data().get(),
        Btiles_rowColIdx.data().get(),
        B_participating_tiles.data().get(), 
        B_participating_tiles_size.data().get(), 
        B_perTileNnz.data().get(), 
        B_d_J.data().get(), 
        B_d_val.data().get(), 
        B_d_rowPtr.data().get(),
        B_rows
    );
    hipEventRecord(B_tileConversion_end, STREAM_B);

    rmm::device_vector<uint16_t> Btiles_transposed_mask(cntB * 16, SPGEMM_STREAM_ALLOCATOR_UINT16(STREAM_B));
    dim3 threads_tBm {256};
    dim3 blocks_tBm {(cntB-1+threads_tBm.x)/threads_tBm.x};
    __transpose_B_mask<<<blocks_tBm, threads_tBm, 0, STREAM_B>>>(Btiles_transposed_mask.data().get(), Btiles.data().get(), cntB);

    // create High level Representation of A -> A_
    rmm::device_vector<int> _A_tileRowPtr_tmp(A_tileRows, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_D));
    rmm::device_vector<int> _A_tileRowPtr(A_tileRows + 1, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_D));
    rmm::device_vector<int> _A_tileColIdx(cntA, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_D));
    rmm::device_vector<float> _A_tileVals(cntA, 1.0f, SPGEMM_STREAM_ALLOCATOR_FLOAT(STREAM_D));
    
    auto newendA = thrust::reduce_by_key(
        ASYNC_EXEC_POLICY(STREAM_D),
        thrust::make_transform_iterator(A_participating_tiles.begin(), getHigh32()),
        thrust::make_transform_iterator(A_participating_tiles.end(), getHigh32()),
        thrust::make_constant_iterator<int>(1),
        _A_tileColIdx.begin(), // borrow _A_tileColIdx as temporary index buffer
        _A_tileRowPtr_tmp.begin()
    );
    thrust::scatter(ASYNC_EXEC_POLICY(STREAM_D), _A_tileRowPtr_tmp.begin(), newendA.second, _A_tileColIdx.begin(), _A_tileRowPtr.begin());
    thrust::exclusive_scan(ASYNC_EXEC_POLICY(STREAM_D), _A_tileRowPtr.begin(), _A_tileRowPtr.end(), _A_tileRowPtr.begin());

    thrust::copy(
        ASYNC_EXEC_POLICY(STREAM_D),
        thrust::make_transform_iterator(A_participating_tiles.begin(), getLow32()),
        thrust::make_transform_iterator(A_participating_tiles.end(), getLow32()),
        _A_tileColIdx.begin()
    );

    // create High level Representation of B -> B_
    rmm::device_vector<int> _B_tileRowPtr_tmp(B_tileRows, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_E));
    rmm::device_vector<int> _B_tileRowPtr(B_tileRows + 1, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_E));
    rmm::device_vector<int> _B_tileColIdx(cntB, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_E));
    rmm::device_vector<float> _B_tileVals(cntB, 1.0f, SPGEMM_STREAM_ALLOCATOR_FLOAT(STREAM_E));
    
    auto newendB = thrust::reduce_by_key(
        ASYNC_EXEC_POLICY(STREAM_E),
        thrust::make_transform_iterator(B_participating_tiles.begin(), getHigh32()),
        thrust::make_transform_iterator(B_participating_tiles.end(), getHigh32()),
        thrust::make_constant_iterator<int>(1),
        _B_tileColIdx.begin(), // borrow _B_tileColIdx as temporary index buffer
        _B_tileRowPtr_tmp.begin()
    );
    thrust::scatter(ASYNC_EXEC_POLICY(STREAM_E), _B_tileRowPtr_tmp.begin(), newendB.second, _B_tileColIdx.begin(), _B_tileRowPtr.begin());
    thrust::exclusive_scan(ASYNC_EXEC_POLICY(STREAM_E), _B_tileRowPtr.begin(), _B_tileRowPtr.end(), _B_tileRowPtr.begin());

    thrust::copy(
        ASYNC_EXEC_POLICY(STREAM_E),
        thrust::make_transform_iterator(B_participating_tiles.begin(), getLow32()),
        thrust::make_transform_iterator(B_participating_tiles.end(), getLow32()),
        _B_tileColIdx.begin()
    );
    
    // transpose _B
    rmm::device_vector<int> _B_tileOffsets(cntB, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_E));
    thrust::sequence(ASYNC_EXEC_POLICY(STREAM_E), _B_tileOffsets.begin(), _B_tileOffsets.end());
    thrust::transform(ASYNC_EXEC_POLICY(STREAM_E), B_participating_tiles.begin(), B_participating_tiles.end(), B_participating_tiles.begin(), swap32());
    {
    auto zit = thrust::make_zip_iterator(thrust::make_tuple(B_participating_tiles.begin(), _B_tileOffsets.begin()));
    thrust::sort(ASYNC_EXEC_POLICY(STREAM_E), zit, zit+B_participating_tiles.size());
    }

    rmm::device_vector<int> _B_tileColPtr_tmp(B_tileCols, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_E));
    rmm::device_vector<int> _B_tileColPtr(B_tileCols + 1, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_E));
    rmm::device_vector<int> _B_tileRowIdx(cntB, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_E));
    auto newendBB = thrust::reduce_by_key(
        ASYNC_EXEC_POLICY(STREAM_E),
        thrust::make_transform_iterator(B_participating_tiles.begin(), getHigh32()),
        thrust::make_transform_iterator(B_participating_tiles.end(), getHigh32()),
        thrust::make_constant_iterator<int>(1),
        _B_tileRowIdx.begin(), // borrow _B_tileRowIdx as temporary index buffer
        _B_tileColPtr_tmp.begin()
    );
    thrust::scatter(ASYNC_EXEC_POLICY(STREAM_E), _B_tileColPtr_tmp.begin(), newendBB.second, _B_tileRowIdx.begin(), _B_tileColPtr.begin());
    thrust::exclusive_scan(ASYNC_EXEC_POLICY(STREAM_E), _B_tileColPtr.begin(), _B_tileColPtr.end(), _B_tileColPtr.begin());
    thrust::copy(
        ASYNC_EXEC_POLICY(STREAM_E),
        thrust::make_transform_iterator(B_participating_tiles.begin(), getLow32()),
        thrust::make_transform_iterator(B_participating_tiles.end(), getLow32()),
        _B_tileRowIdx.begin()
    );

    hipStreamSynchronize(STREAM_D);
    rmm::device_vector<long long>().swap(A_participating_tiles);
    rmm::device_vector<int>().swap(_A_tileRowPtr_tmp);
    hipStreamSynchronize(STREAM_E);
    rmm::device_vector<long long>().swap(B_participating_tiles);
    rmm::device_vector<int>().swap(_B_tileRowPtr_tmp);

    rmm::device_vector<int> 
    _C_tilePtr(SPGEMM_STREAM_ALLOCATOR_INT(STREAM_C)), 
    _C_tileColIdx(SPGEMM_STREAM_ALLOCATOR_INT(STREAM_C)),
    _C_tileRowIdx(SPGEMM_STREAM_ALLOCATOR_INT(STREAM_C));

    hipEvent_t accumulator_start;
    hipEvent_t accumulator_end;
    hipEventCreate(&accumulator_start);
    hipEventCreate(&accumulator_end);

    auto pem_spgemm_start = std::chrono::high_resolution_clock::now();
    cusparse_highLevelMultiply
    (
        _A_tileRowPtr.data().get(),
        _A_tileColIdx.data().get(),
        _A_tileVals.data().get(),
        A_tileRows,
        A_tileCols,
        _A_tileVals.size(),
        _B_tileRowPtr.data().get(),
        _B_tileColIdx.data().get(),
        _B_tileVals.data().get(),
        B_tileRows,
        B_tileCols,
        _B_tileVals.size(),
        &_C_tilePtr, &_C_tileColIdx, &_C_tileRowIdx,
        STREAM_C,
        ASYNC_EXEC_POLICY(STREAM_C)
    );
    
    std::cout << "\nCOUNTING PAIRS\n";

    dim3 threads_sp{256};
    dim3 blocks_sp{(_C_tilePtr.back()-1+threads_sp.x)/threads_sp.x};
    
    int *d_pairs_count, *h_pairs_count;
    hipMallocAsync(&d_pairs_count, sizeof(int), STREAM_C);
    h_pairs_count = new int;
    rmm::device_vector<int> pairs_insertion_offset(_C_tilePtr.back()+1, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_C));
    search_pairs<0><<<blocks_sp, threads_sp, 0, STREAM_C>>>
    (
        nullptr,
        nullptr,
        nullptr,
        _C_tilePtr.data().get(),
        _C_tileColIdx.data().get(),
        _A_tileRowPtr.data().get(),
        _A_tileColIdx.data().get(),
        _B_tileColPtr.data().get(),
        _B_tileRowIdx.data().get(),
        _C_tilePtr.size()-1,
        _C_tileColIdx.size(),
        _B_tileOffsets.data().get(),
        _C_tileRowIdx.data().get(),
        d_pairs_count,
        pairs_insertion_offset.data().get()
    );
    hipMemcpyAsync(h_pairs_count, d_pairs_count, sizeof(int), hipMemcpyDeviceToHost, STREAM_C);
    hipStreamSynchronize(STREAM_C);
    std::cout << "Pairs count: " << *h_pairs_count << "\n";

    thrust::exclusive_scan(ASYNC_EXEC_POLICY(STREAM_C), pairs_insertion_offset.begin(), pairs_insertion_offset.end(), pairs_insertion_offset.begin());

    rmm::device_vector<int> d_pairs_a(*h_pairs_count);
    rmm::device_vector<int> d_pairs_b(*h_pairs_count);
    rmm::device_vector<int> C_targetTile(*h_pairs_count);
    search_pairs<1><<<blocks_sp, threads_sp, 0, STREAM_C>>>
    (
        d_pairs_a.data().get(),
        d_pairs_b.data().get(),
        C_targetTile.data().get(),
        _C_tilePtr.data().get(),
        _C_tileColIdx.data().get(),
        _A_tileRowPtr.data().get(),
        _A_tileColIdx.data().get(),
        _B_tileColPtr.data().get(),
        _B_tileRowIdx.data().get(),
        _C_tilePtr.size()-1,
        _C_tileColIdx.size(),
        _B_tileOffsets.data().get(),
        _C_tileRowIdx.data().get(),
        d_pairs_count,
        pairs_insertion_offset.data().get()
    );

    rmm::device_vector<TileCSR_C_rev<ValueType>> Ctiles(_C_tileColIdx.size(), SPGEMM_STREAM_ALLOCATOR_TILECSRC_REV(STREAM_C));
    rmm::device_vector<int> _C_perTileNnz(_C_tileColIdx.size() + 1, SPGEMM_STREAM_ALLOCATOR_INT(STREAM_C));

    int numBlocksPerSm_aC = 0;
    int numThreads_aC = 128;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm_aC, (void*)allocate_C<ValueType>, numThreads_aC, 0);
    std::cout << "allocate_C Max block count per SM with 128 threads: " << numBlocksPerSm_aC << "\n";

    auto args1 = d_pairs_a.data().get();
    auto args11 = d_pairs_b.data().get();
    auto args2 = d_pairs_a.size();
    auto args3 =  Ctiles.data().get();
    auto args4 = Ctiles.size();
    auto args5 = _C_perTileNnz.data().get();
    auto args6 = C_targetTile.data().get();
    auto args7 =  Atiles.data().get();
    auto args8 =  Btiles.data().get();
    auto args88 = Btiles_transposed_mask.data().get();

    void *args[] = {
        static_cast<void*>(&args1),
        static_cast<void*>(&args11),
        static_cast<void*>(&args2),
        static_cast<void*>(&args3),
        static_cast<void*>(&args4),
        static_cast<void*>(&args5),
        static_cast<void*>(&args6),
        static_cast<void*>(&args7),
        static_cast<void*>(&args8),
        static_cast<void*>(&args88)
    };


    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    dim3 threads_aC {numThreads_aC};
    dim3 blocks_aC {numBlocksPerSm_aC * deviceProp.multiProcessorCount};
    CHECK_CUDA( hipLaunchCooperativeKernel((void*)allocate_C<ValueType>, blocks_aC, threads_aC, args, 0, STREAM_C) )

    thrust::exclusive_scan(ASYNC_EXEC_POLICY(STREAM_C), _C_perTileNnz.begin(), _C_perTileNnz.end(), _C_perTileNnz.begin());

    rmm::device_vector<ValueType> Ctiles_vals(_C_perTileNnz.back(), SPGEMM_STREAM_ALLOCATOR_VALUETYPE(STREAM_C));
    rmm::device_vector<uint8_t> Ctiles_rowColIdx(_C_perTileNnz.back(), SPGEMM_STREAM_ALLOCATOR_UINT8(STREAM_C));

    int numBlocksPerSm_Cs = 0;
    int numThreads_Cs = 128;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm_Cs, (void*)C_setOffsets<ValueType>, numThreads_Cs, 0);
    std::cout << "C_setOffsets Max block count per SM with 128 threads: " << numBlocksPerSm_Cs << "\n";

    auto args9 = Ctiles_vals.data().get();
    auto args10 = Ctiles_rowColIdx.data().get();
    void *args_2[] = {
        static_cast<void*>(&args3),
        static_cast<void*>(&args4),
        static_cast<void*>(&args5),
        static_cast<void*>(&args9),
        static_cast<void*>(&args10)
    };

    dim3 threads_Cs {numThreads_Cs};
    dim3 blocks_Cs {numBlocksPerSm_Cs * deviceProp.multiProcessorCount};
    CHECK_CUDA( hipLaunchCooperativeKernel((void*)C_setOffsets<ValueType>, blocks_Cs, threads_Cs, args_2, 0, STREAM_C) );

    std::cout << "\nACCUMULATOR PHASE\n\n\n";

    dim3 threads_mp {128};
    dim3 blocks_mp {((Ctiles.size()-1+threads_mp.x/threads_mp.x)+3)/4};

    hipEventRecord(accumulator_start, STREAM_C);
    multiply_pairs_default<ValueType><<<blocks_mp, threads_mp, 0, STREAM_C>>>
    (
        d_pairs_a.data().get(),
        d_pairs_b.data().get(),
        Ctiles.data().get(),
        Ctiles.size(),
        _C_perTileNnz.data().get(),
        C_targetTile.data().get(),
        Atiles.data().get(),
        Btiles.data().get(),
        Btiles_transposed_mask.data().get(),
        pairs_insertion_offset.data().get()
    );
    hipEventRecord(accumulator_end, STREAM_C);

    hipDeviceSynchronize();
    auto pem_spgemm_end = std::chrono::high_resolution_clock::now();
    auto pem_spgemm_duration = std::chrono::duration<double, std::milli>(pem_spgemm_end-pem_spgemm_start);
    
    float Aconversion, Bconversion;
    hipEventElapsedTime(&Aconversion, A_tileConversion_start, A_tileConversion_end);
    hipEventElapsedTime(&Bconversion, B_tileConversion_start, B_tileConversion_end);

    float accumulator_time;
    hipEventElapsedTime(&accumulator_time, accumulator_start, accumulator_end);
    
    std::cout << "<---Program done--->\n";
    std::cout << "Matrix A CSR to tile conversion took " << Aconversion << " ms\n";
    std::cout << "Matrix B CSR to tile conversion took " << Bconversion << " ms\n";
    std::cout << "Accumulator took " << accumulator_time << " ms\n";
    std::cout << "PEM-SPGEMM took " << std::fixed << std::setprecision(2) << pem_spgemm_duration.count() << " ms\n";
    std::cout << "C tiles: " << Ctiles.size() << "\n";
    std::cout << "C nnz: " << _C_perTileNnz.back() << "\n";

    if(!atoi(argv[2]))
    {
        std::cout << "Not saving results. Exiting.\n";
        std::atexit([]{hipDeviceReset();});
        return 0;
    }

    rmm::device_vector<int> Crows(_C_perTileNnz.back(), SPGEMM_STREAM_ALLOCATOR_INT(STREAM_C));
    rmm::device_vector<int> Ccols(_C_perTileNnz.back(), SPGEMM_STREAM_ALLOCATOR_INT(STREAM_C));
    rmm::device_vector<ValueType> Cvals(_C_perTileNnz.back(), SPGEMM_STREAM_ALLOCATOR_VALUETYPE(STREAM_C));

    dim3 threads_sC {tileSize * tileSize};
    dim3 blocks_sC {(Ctiles.size()+7)/8};

    sanitize_C<<<blocks_sC, threads_sC, 0, STREAM_C>>>
    (
        Crows.data().get(), 
        Ccols.data().get(), 
        Cvals.data().get(), 
        Ctiles.data().get(), 
        Ctiles.size(),
        _C_tilePtr.data().get(),
        _C_tilePtr.size(), 
        _C_tileRowIdx.data().get(),
        _C_tileColIdx.data().get(), 
        _C_perTileNnz.data().get()
    );

    {
    auto zit = thrust::make_zip_iterator(Crows.begin(), Ccols.begin(), Cvals.begin());
    thrust::stable_sort(ASYNC_EXEC_POLICY(STREAM_C), zit, zit + _C_perTileNnz.back());
    }

    std::cout << "Saving results to /tmp/SPGEMM_RESULT_*.txt\n";

    char const *filename0 = "/tmp/SPGEMM_RESULT_NNZ.txt";
    char const *filename1 = "/tmp/SPGEMM_RESULT_ROWS.txt";
    char const *filename2 = "/tmp/SPGEMM_RESULT_COLS.txt";
    char const *filename3 = "/tmp/SPGEMM_RESULT_VALS.txt";
    char const *filename4 = "/tmp/SPGEMM_RESULT_COMBINE.txt";

    std::ofstream outfile;

    auto print = [&outfile] (auto v) { outfile << v << "\n"; };
    auto print2 = [&outfile] (auto v) { outfile << v << " "; };
    
    thrustHvec<int> hCrows(Crows.size());
    thrustHvec<int> hCcols(Ccols.size());
    thrustHvec<ValueType> hCvals(Cvals.size());

    hipStreamSynchronize(STREAM_C);
    thrust::copy(Crows.begin(), Crows.end(), hCrows.begin());
    thrust::copy(Ccols.begin(), Ccols.end(), hCcols.begin());
    thrust::copy(Cvals.begin(), Cvals.end(), hCvals.begin());

    outfile.open(filename4, std::ios::out);
    outfile << "Rows\n";
    std::for_each(hCrows.begin(), hCrows.end(), print2);
    outfile << "\nCols\n";
    std::for_each(hCcols.begin(), hCcols.end(), print2);
    outfile << "\nVals\n";
    outfile << std::fixed << std::setprecision(std::numeric_limits<double>::max_digits10);
    std::for_each(hCvals.begin(), hCvals.end(), print2);
    outfile.close();

    outfile.open(filename0, std::ios::out);
    outfile << _C_perTileNnz.back();
    outfile.close();

    outfile.open(filename1, std::ios::out);
    std::for_each(hCrows.begin(), hCrows.end(), print);
    outfile.close();

    outfile.open(filename2, std::ios::out);
    std::for_each(hCcols.begin(), hCcols.end(), print);
    outfile.close();

    outfile.open(filename3, std::ios::out);
    outfile << std::fixed << std::setprecision(std::numeric_limits<double>::max_digits10);
    std::for_each(hCvals.begin(), hCvals.end(), print);
    outfile.close();


    hipEventDestroy(A_tileConversion_start);
    hipEventDestroy(A_tileConversion_end);
    hipEventDestroy(B_tileConversion_start);
    hipEventDestroy(B_tileConversion_end);

    std::atexit([]{hipDeviceReset();});
    return 0; // <--------------------------------------------------------------------------------------------------------------------------------

    // streams are destroyed by rmm
}
